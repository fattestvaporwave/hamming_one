
#include <hip/hip_runtime.h>
#include <cstdint>
#include <random>
#include <stdlib.h>
#include <time.h>

#define _POSIX_C_SOURCE 199309L
#define COUNT 5000

#ifdef __DRIVER_TYPES_H__
    #ifndef DEVICE_RESET
        #define DEVICE_RESET hipDeviceReset();
    #endif
#else
    #ifndef DEVICE_RESET
        #define DEVICE_RESET 
    #endif
#endif

#define checkCudaErrors(val) { check((val), __LINE__); }

void check(hipError_t cudaStatus, int line)
{
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA Error %d at line %d: %s\n", line, cudaStatus, hipGetErrorString(cudaStatus));
        DEVICE_RESET
            exit(EXIT_FAILURE);
    }
}

std::random_device rd;
std::mt19937_64 gen(rd());
std::uniform_int_distribution<unsigned long long> dis(
    std::numeric_limits<std::uint64_t>::min(),
    std::numeric_limits<std::uint64_t>::max()
);

void generateSeqs(uint64_t* seqs) 
{
    for (uint64_t i = 0; i < COUNT * 8; i++) {
        seqs[i] = (dis(gen) >> 63) & UINT64_MAX;
    }
}

void printSeqs(uint64_t* seqs) 
{
    for (long i = 0; i < COUNT * 8; i += 8)
    {
        printf("Sequence %li:  ", i / 8 + 1);
        for (long j = 0; j < 8; j++)
        {
            printf("%I64i ", seqs[i + j]);
        }
        printf("\n");
    }
}

//-------------------CPU SOLUTION------------------------

int hammingDistance(uint64_t n1, uint64_t n2)
{
    uint64_t x = n1 ^ n2;
    long setBits = 0;

    while (x > 0) {
        setBits += x & 1;
        x >>= 1;
    }

    return setBits;
}

void hamming(const uint64_t* seqs, bool* pairs) 
{
    long distance;

    for (long i = 0; i < COUNT; i++) 
    {
        for (long j = i + 1; j < COUNT; j++) 
        {
            distance = 0;

            for (long k = 0; k < 8; k++)
                distance += hammingDistance(seqs[i * 8 + k], seqs[j * 8 + k]);

            if (distance == 1)
                pairs[i * COUNT + j] = true;
        }
    }
}

//-------------------GPU SOLUTION------------------------

__device__ int hammingDistanceCuda(uint64_t n1, uint64_t n2)
{
    uint64_t x = n1 ^ n2;
    long setBits = 0;

    while (x > 0) 
    {
        setBits += x & 1;
        x >>= 1;
    }

    return setBits;
}

__global__ void isHammingOneCuda(const uint64_t* seqs, bool* pairs) 
{
    unsigned int threadsPerBlock = blockDim.x * blockDim.y;
    unsigned int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
    unsigned int blockNumInGrid = blockIdx.x + gridDim.x * blockIdx.y;

    unsigned int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

    uint64_t comparedSeq[8];
    for (unsigned int i = 0; i < 8; i++) 
        comparedSeq[i] = seqs[globalThreadNum * 8 + i];

    for (unsigned int i = globalThreadNum + 1; i < COUNT; i++)
    {
        long distance = 0;

        for (unsigned int j = 0; j < 8; j++) 
            distance += hammingDistanceCuda(comparedSeq[j], seqs[i * 8 + j]);

        if (distance == 1) 
            pairs[globalThreadNum * COUNT + i] = true;
    }
}

hipError_t hammingCuda(const uint64_t* seqs, bool* pairs) 
{
    dim3 block(32, 4);
    dim3 grid(block.x * block.y, ceil((double)COUNT / (block.x * block.y)));

    isHammingOneCuda <<<grid, block>>> (seqs, pairs);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    return hipSuccess;
}

//-------------------MAIN------------------------

int main() 
{
    uint64_t *seqs;
    bool *pairs;
    clock_t start, finish;
    double duration;
    uint64_t counter = 0;

    hipMallocManaged(&seqs, COUNT * 8 * sizeof(uint64_t));
    hipMallocManaged(&pairs, COUNT * COUNT * sizeof(bool));
    generateSeqs(seqs);
    printSeqs(seqs);

    for (long i = 0; i < COUNT * COUNT; i++)
        pairs[i] = false;

    printf("-------------CPU SOLUTION------------\n");
    start = clock();
    hamming(seqs, pairs);
    finish = clock();
    duration = (double)(finish - start) / CLOCKS_PER_SEC;
    printf("Time:  %2.3f seconds\n", duration);
    
    for (uint64_t i = 0; i < COUNT * COUNT; i++)
        if (pairs[i])
            counter++;
    printf("Pairs with Hamming distance of 1:  %I64i\n", counter);
    
    counter = 0;
    for (long i = 0; i < COUNT * COUNT; i++)
        pairs[i] = false;

    printf("\n-------------GPU SOLUTION------------\n");
    start = clock();
    hammingCuda(seqs, pairs);
    finish = clock();
    duration = (double)(finish - start) / CLOCKS_PER_SEC;
    printf("Time:  %2.3f seconds\n", duration);

    for (uint64_t i = 0; i < COUNT * COUNT; i++)
        if (pairs[i])
            counter++;
    printf("Pairs with Hamming distance of 1:  %I64i\n", counter);

    checkCudaErrors(hipFree(seqs));
    checkCudaErrors(hipFree(pairs));
	getchar();
    return 0;
}
